#include <iostream>
#include <hip/hip_runtime.h>

#define N 16  // Size of the matrices

// CUDA kernel for matrix multiplication
__global__ void matrixMulCUDA(float *C, const float *A, const float *B, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if(row < width && col < width) {
        for(int k = 0; k < width; ++k) {
            sum += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);
    float h_A[N*N], h_B[N*N], h_C[N*N];

    // Initialize matrices
    for(int i = 0; i < N*N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i % N);
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the kernel
    matrixMulCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_A, d_B, N);

    // Copy the result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Display a portion of the result matrix
    std::cout << "Result matrix C:" << std::endl;
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

